//Cuda version of dijkstras

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//Size of the graph.
//Change to user input later
#define MAX_SIZE 1000

// //Kernel Code Goes Here
__global__ void findNeighbors(int *graph, int source, int *isPermanent, int *vertices, int *cost, int vertexCount)
{
	long index = threadIdx.x + blockIdx.x * blockDim.x;
	if(graph[(source - 1) * vertexCount + index] != 0)
	{
		if(vertices[index] != 0 && (cost[index] > graph[(source - 1) * vertexCount + index] + cost[source - 1]))
		{
			cost[index] = graph[(source - 1) * vertexCount + index] + cost[source - 1];
		}
		else if(vertices[index] == 0)
		{
			vertices[index] = index + 1;
			cost[index] = cost[source - 1] + graph[(source - 1) * vertexCount + index];
		}
	}
}

//function declarations
int find_minimal_distance(int *vertices, int *cost, int *isPermanent);
int checked(int *vertices, int *cost, int *isPermanent);
void path(int (*graph)[MAX_SIZE], int *vertices, int *cost, int current, int **paths);

//Host Code
int main(int argc, char *argv[])
{
	//variable declarations
	//===============================================================================
	

	int i, k, j, v1, v2, w, flag = 0; //indicies 
	int tempGraph[MAX_SIZE][MAX_SIZE] = {0};
	float timeResult;

	int *graph, *vertices, *cost, *isPermanent;;
	int *d_vertices, *d_cost, *d_isPermanent, *d_graph;
	int **paths;
	paths = (int **)malloc(sizeof(int*) * MAX_SIZE);

	for(i = 0; i < MAX_SIZE; i++)
	{
		paths[i] = (int *)malloc(sizeof(int) * MAX_SIZE);
	}
	int currentVertice;
	long size = sizeof(int) * MAX_SIZE;

	graph = (int *)malloc(sizeof(int) * MAX_SIZE * MAX_SIZE);
	vertices = (int *)malloc(size);
	cost = (int *)malloc(size);
	isPermanent = (int *)malloc(size);

	hipMalloc((void **) &d_graph, sizeof(int) * MAX_SIZE * MAX_SIZE);
	hipMalloc((void **) &d_vertices, size);
	hipMalloc((void **) &d_cost, size);
	hipMalloc((void **) &d_isPermanent, size);

	//Initialize the timers to be used in the cuda part.
	hipEvent_t startTime, stopTime;
	hipEventCreate(&startTime);
	hipEventCreate(&stopTime);


	//Get the data
	//================================================================================
	//The graph will be read from a file
	FILE *graph_source = fopen("bigGraph", "r");
	FILE *results = fopen("cResults.dat", "w");

	//Check if file has contents
	if(graph_source == NULL)
	{
		printf("The file could not be opened.\n");
	}

	//Get all the vertices from the file and add them to the adyecency matrix
	// while(fscanf(graph_source, "%i %i %i",&v1 ,&v2 ,&w) != EOF)
	// {
	// 	tempGraph[v1 - 1][v2 - 1] = w;
	// }

	for(i = 0; i < MAX_SIZE; i++) 
	{
		for(j = 0; j < MAX_SIZE; j++) 
		{
			fscanf(graph_source, "%d", &tempGraph[i][j]);
		}
	}

	//Fill up the temp array
	for(i = 0; i < MAX_SIZE; i++)
	{
		cost[i] = 0;
		isPermanent[i] = 0;
		for(k = 0; k < MAX_SIZE; k++)
		{
			graph[i * MAX_SIZE + k] = tempGraph[i][k]; 
		}
	}
	//================================================================================

	//User can choose which vertice to assign as source
	int source;
	printf("Source: \n");
	scanf("%d", &source);

	currentVertice = source;
	cost[source - 1] = 0;
	isPermanent[source - 1] = 1;
	vertices[source - 1] = source;

	
	hipMemcpy(d_graph, graph, sizeof(int) * MAX_SIZE * MAX_SIZE, hipMemcpyHostToDevice);
	
	//Start timer
	hipEventRecord(startTime, 0);

	//Calculate Shortest Path
	//==================================================================================
	while(flag == 0)
	{
		//Copy the arrays to the device;
		hipMemcpy(d_vertices, vertices, size, hipMemcpyHostToDevice);
		hipMemcpy(d_cost, cost, size, hipMemcpyHostToDevice);
		hipMemcpy(d_isPermanent, isPermanent, size, hipMemcpyHostToDevice);

		findNeighbors<<<1,MAX_SIZE>>>(d_graph, currentVertice, d_isPermanent, d_vertices, d_cost, MAX_SIZE);

		//Copy the arrays back from the device, this is the time consuming part
		hipMemcpy(vertices, d_vertices, size, hipMemcpyDeviceToHost);
		hipMemcpy(cost, d_cost, size, hipMemcpyDeviceToHost);
		hipMemcpy(isPermanent, d_isPermanent, size, hipMemcpyDeviceToHost);

		currentVertice = find_minimal_distance(vertices, cost, isPermanent);

		isPermanent[currentVertice - 1] = 1;

		flag = checked(vertices, cost, isPermanent);
		path(tempGraph, vertices, cost, currentVertice, paths);
	}
	//Results
	//======================================================================================

	//End timer
	hipEventRecord(stopTime, 0);

	hipEventSynchronize(startTime);
	hipEventSynchronize(stopTime);

	hipEventElapsedTime(&timeResult, startTime, stopTime);
	fprintf(results, "Time elapsed for CUDA Dijkstras: %f milliseconds\n", timeResult);

	for(i = 0; i < MAX_SIZE; i++)
	{
		fprintf(results, "Vertice: %d Distance: %d Path: ", vertices[i], cost[i]);
		for(k = 0; k < MAX_SIZE; k++)
		{
			if(paths[i][k] != 0)
				fprintf(results, "%d ", paths[i][k]);
		}
		fprintf(results,"\n");
	}

	//Sanitize
	//==================================================================================
	//Close the file
	free(vertices);
	free(isPermanent);
	free(cost);
	free(graph);

	for(i = 0; i < MAX_SIZE; i++)
	{
		free(paths[i]);
	}
	free(paths);
	fclose(results);
	fclose(graph_source);

	hipFree(d_graph);
	hipFree(d_cost);
	hipFree(d_isPermanent);
	hipFree(d_vertices);

}
int find_minimal_distance(int *vertices, int *cost, int *isPermanent)
{
	int k, j;

	int tempVertice;
	//Find the first vertice that is not permanent and hold it in temp
		for(k = 0; k < MAX_SIZE; k++)
		{
			if(isPermanent[k] == 0 && cost[k] > 0)
			{
				tempVertice = vertices[k];
				break;
			}
		}

		//Find the smallest amongsts the temporary distances. 
		for(j = 0; j < MAX_SIZE; j++)
		{
			
			if(isPermanent[j] == 0 && (cost[j] < cost[tempVertice - 1] && cost[j] != 0))
			{

				tempVertice = vertices[j];
			}
		}
		return tempVertice;
}

void path(int (*graph)[MAX_SIZE], int *vertices, int *cost, int current, int **paths)
{
	//variables used
	int w, c = 0, origin = current - 1;
	//If the vertex connects to nowhere, don't check for the path.
	if(cost[current - 1] == 0)
	{
		return;
	}
	//The first position is occupied by the currentVertice itself
	while(cost[current - 1]!= 0)
	{
		for(w = 0; w < MAX_SIZE; w++)
		{
			if(graph[current - 1][w] != 0)
			{
				//If the minimal distance of the current node minus the the weight of one of it's connected paths
				//is equal to the minimal distance of the vertice t the other side of the path
				//That is the correct path.
				if(cost[current - 1] - graph[current - 1][w] == cost[w])
				{
					paths[origin][c] = current;;
					current = vertices[w];
					c++;
					break;
				}
			}
		}
	}
	paths[origin][c] = current;
}

int checked(int *vertices, int *cost, int *isPermanent)
{
	int b;
	for(b = 0; b < MAX_SIZE; b++)
	{
		//If there is at least one that is not permanent, break
		if(isPermanent[b] == 0)
		{
			break;
		}
	}
	if(b == MAX_SIZE)
	{
		return 1;
	}
	else
	{
		return 0;
	}
}


